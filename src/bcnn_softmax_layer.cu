
#include <hip/hip_runtime.h>
/*
* Copyright (c) 2016 Jean-Noel Braun.
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/


#ifdef BCNN_USE_CUDA

#include "bcnn/bcnn.h"

__global__ void _bcnn_forward_softmax_layer_kernel(int n, int batch, float *input, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    int b = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    
    if (b >= batch) {
        return;
    }
    
    for (i = 0; i < n; ++i) {
        int val = input[i+b*n];
        largest = (val>largest) ? val : largest;
    }

    for (i = 0; i < n; ++i) {
        sum += exp(input[i+b*n]-largest);
    }

    sum = (sum != 0) ? largest+log(sum) : largest-100;

    for (i = 0; i < n; ++i) {
        output[i+b*n] = exp(input[i+b*n]-sum);
    }
}

int bcnn_forward_softmax_layer_gpu(bcnn_connection *conn)
{
    int src_size = conn->src_tensor.w * conn->src_tensor.h * conn->src_tensor.c;
    int batch_size = conn->dst_tensor.b;
    bcnn_tensor src = conn->src_tensor;
    bcnn_tensor dst = conn->dst_tensor;

    _bcnn_forward_softmax_layer_kernel<<<bcnn_cuda_gridsize(batch_size), BCNN_CUDA_THREADS>>>(src_size,
        batch_size, src.data_gpu, dst.data_gpu);
    bcnn_cuda_check(cudaPeekAtLastError());

    return BCNN_SUCCESS;
}

int bcnn_backward_softmax_layer_gpu(bcnn_connection *conn)
{
    int size = conn->src_tensor.w * conn->src_tensor.h * conn->src_tensor.c
        * conn->dst_tensor.b;
    bcnn_tensor src = conn->src_tensor;
    bcnn_tensor dst = conn->dst_tensor;

    bcnn_cuda_axpy(size, 1, dst.grad_data_gpu, 1, src.grad_data_gpu, 1);

    return BCNN_SUCCESS;
}


#endif